
#include <hip/hip_runtime.h>


#if defined(TB_CUTE)
//    #include <complex>
    #include <contract/contract.h>
    #include <tools/class_tic_toc.h>
    #include <tools/log.h>
    #include <tools/prof.h>

    #include <stdio.h>
    #include <stdlib.h>

    #include <cuda_runtime.h>
    #include <cutensor.h>

    #include <unordered_map>
    #include <vector>

    // Handle cuTENSOR errors
    #define HANDLE_ERROR(x)                                                                                                                                    \
        {                                                                                                                                                      \
            const auto err = x;                                                                                                                                \
            if(err != CUTENSOR_STATUS_SUCCESS) {                                                                                                               \
                tools::log->critical("{} in line {}", cutensorGetErrorString(err), __LINE__);                                                                  \
                exit(err);                                                                                                                                     \
            }                                                                                                                                                  \
        }

    #define HANDLE_CUDA_ERROR(x)                                                        \
    { const auto err = x;                                                               \
      if( err != cudaSuccess )                                                          \
      {  tools::log->critical("{} in line {}", cudaGetErrorString(err),__LINE__);       \
                exit(err);                                                              \
      }                                                                                 \
    }


long get_ops_m(long chiL, long chiR, long d, long m) {
    long step1   = chiL * chiL * chiR * m * m * d;
    long step2_m = chiL * chiR * d * d * m * m * (d * m + 1);
    long step3_m = chiL * chiR * d * d * m * m * (d * m + 1);
    long step4   = chiL * chiR * d * m * (chiR * m * m * m + m * m + 1);
    return step1 + step2_m + step3_m + step4;
}


template<typename Scalar>
class Meta {
    private:
    Scalar *      d_ptr    = nullptr;
    const Scalar *h_ptr   = nullptr;

    public:
    using value_type = Scalar;

    std::vector<int>     mode;
    std::vector<int64_t> extent;

    template<auto rank>
    Meta(const Eigen::Tensor<Scalar, rank> &tensor, const std::vector<int> &mode_) : h_ptr(tensor.data()), mode(mode_) {
        if(rank != mode.size()) throw std::runtime_error("Rank mismatch");
        for(size_t idx = 0; idx < rank; idx++) extent.push_back(tensor.dimension(idx));
    }

    Meta(const std::vector<int> &mode_, const std::vector<int64_t> & extent_) : mode(mode_),extent(extent_) {
        if(mode.size() != extent.size()) throw std::runtime_error("Mode and extent size mismatch");
    }

    ~Meta() {
//        if(d_ptr) HANDLE_CUDA_ERROR(cudaFree(d_ptr));

    }

    auto size() {
        size_t size = 1;
        for(auto &ext : extent) size *= ext;
        return size;
    }

    size_t   byteSize() { return size() * sizeof(Scalar); }
    uint32_t rank() { return mode.size(); }
    Scalar * data_d() { return d_ptr; }
    const Scalar * data_h() {
        return h_ptr;
    }
    void copyToDevice() {
        if(data_d() == nullptr) HANDLE_CUDA_ERROR(cudaMalloc((void **) &d_ptr, byteSize()));
        if(data_h() == nullptr) return; // Nothing to copy
        HANDLE_CUDA_ERROR(cudaMemcpy(data_d(), data_h(), byteSize(), cudaMemcpyHostToDevice));
    }

    void copyFromDevice(Scalar * data_h) {
        if(data_h == nullptr) throw std::runtime_error("Cannot copy from device: Host data is null");
        if(data_d() == nullptr) return; // Nothing to copy
        HANDLE_CUDA_ERROR(cudaMemcpy(data_h, data_d(), byteSize(), cudaMemcpyDeviceToHost));
    }

    void free(){
        if(d_ptr) {
            HANDLE_CUDA_ERROR(cudaFree(d_ptr));
        }
    }
};

template<typename Scalar>
void cuTensorContract(Meta<Scalar> &tensor_R, Meta<Scalar> &tensor_A, Meta<Scalar> &tensor_B) {
    // CUDA types
    cudaDataType_t        typeCutensor;
    cutensorComputeType_t typeCompute;
    if constexpr(std::is_same_v<Scalar, double>) {
        typeCutensor = CUDA_R_64F;
        typeCompute  = CUTENSOR_COMPUTE_64F;
        tools::log->trace("Detected type fp64");
    } else if(std::is_same_v<Scalar, float>) {
        typeCutensor = CUDA_R_32F;
        typeCompute  = CUTENSOR_COMPUTE_32F;
        tools::log->trace("Detected type fp32");
    } else
        throw std::runtime_error("Wrong type selected");

    Scalar alpha = 1.0;
    Scalar beta  = 0.0;

    // Initialize cuTENSOR library
    cutensorHandle_t handle;
    cutensorInit(&handle);

    // Create Tensor Descriptors
    cutensorTensorDescriptor_t desc_A;
    cutensorTensorDescriptor_t desc_B;
    cutensorTensorDescriptor_t desc_R;
    HANDLE_ERROR(cutensorInitTensorDescriptor(&handle, &desc_A, tensor_A.rank(), tensor_A.extent.data(), nullptr, typeCutensor, CUTENSOR_OP_IDENTITY));
    HANDLE_ERROR(cutensorInitTensorDescriptor(&handle, &desc_B, tensor_B.rank(), tensor_B.extent.data(), nullptr, typeCutensor, CUTENSOR_OP_IDENTITY));
    HANDLE_ERROR(cutensorInitTensorDescriptor(&handle, &desc_R, tensor_R.rank(), tensor_R.extent.data(), nullptr, typeCutensor, CUTENSOR_OP_IDENTITY));

    tools::log->trace("Initialize cuTENSOR and tensor descriptors");

    // Retrieve the memory alignment for each tensor
    uint32_t alignmentRequirement_A;
    uint32_t alignmentRequirement_B;
    uint32_t alignmentRequirement_R;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(&handle, tensor_A.data_d(), &desc_A, &alignmentRequirement_A));
    HANDLE_ERROR(cutensorGetAlignmentRequirement(&handle, tensor_B.data_d(), &desc_B, &alignmentRequirement_B));
    HANDLE_ERROR(cutensorGetAlignmentRequirement(&handle, tensor_R.data_d(), &desc_R, &alignmentRequirement_R));

    tools::log->trace("Query best alignment requirement for our pointers");

    // Create the Contraction Descriptor
    cutensorContractionDescriptor_t desc;
    HANDLE_ERROR(cutensorInitContractionDescriptor(&handle, &desc, &desc_A, tensor_A.mode.data(), alignmentRequirement_A, &desc_B, tensor_B.mode.data(),
                                                   alignmentRequirement_B, &desc_R, tensor_R.mode.data(), alignmentRequirement_R, &desc_R, tensor_R.mode.data(),
                                                   alignmentRequirement_R, typeCompute));

    tools::log->trace("Initialize contraction descriptor");

    /* ***************************** */

    // Set the algorithm to use
    cutensorContractionFind_t find;
    HANDLE_ERROR(cutensorInitContractionFind(&handle, &find, CUTENSOR_ALGO_DEFAULT));

    tools::log->trace("Initialize settings to find algorithm");

    /* ***************************** */

    // Query workspace
    size_t worksize = 0;
    HANDLE_ERROR(cutensorContractionGetWorkspace(&handle, &desc, &find, CUTENSOR_WORKSPACE_RECOMMENDED, &worksize));

    // Allocate workspace
    void *work = nullptr;
    if(worksize > 0) {
        if(cudaSuccess != cudaMalloc(&work, worksize)) // This is optional!
        {
            work     = nullptr;
            worksize = 0;
        }
    }

    tools::log->trace("Query recommended workspace size and allocate it");

    /* ***************************** */

    // Create Contraction Plan
    cutensorContractionPlan_t plan;
    HANDLE_ERROR(cutensorInitContractionPlan(&handle, &plan, &desc, &find, worksize));

    tools::log->trace("Create plan for contraction");

    /* ***************************** */

    cutensorStatus_t err;
    cudaStream_t stream = nullptr;
    // Execute the tensor contraction
    err = cutensorContraction(&handle, &plan, (void *) &alpha, tensor_A.data_d(), tensor_B.data_d(), (void *) &beta, tensor_R.data_d(), tensor_R.data_d(), work,
                              worksize, stream);
    cudaDeviceSynchronize();

    // Check for errors
    if(err != CUTENSOR_STATUS_SUCCESS) {  tools::log->error("{}", cutensorGetErrorString(err)); }

    tools::log->trace("Execute contraction from plan");

    if(work) cudaFree(work);

    tools::log->trace("Successful completion");
}

template<typename Scalar>
Eigen::Tensor<Scalar, 3> contract::hamiltonian_squared_dot_psi_cute(const Eigen::Tensor<Scalar, 3> &psi_in, const Eigen::Tensor<Scalar, 4> &mpo,
                                                                    const Eigen::Tensor<Scalar, 4> &envL, const Eigen::Tensor<Scalar, 4> &envR) {
    tools::prof::t_ham_sq_psi_cute->tic();
    Eigen::DSizes<long, 3>   dsizes = psi_in.dimensions();
    Eigen::Tensor<Scalar, 3> ham_sq_psi(dsizes);
    Eigen::Tensor<Scalar, 3> psi_shuffled = psi_in.shuffle(Textra::array3{1, 0, 2});

    // Extents
    std::unordered_map<int, int64_t> ext;
    ext['i'] = psi_shuffled.dimension(0);
    ext['j'] = psi_shuffled.dimension(1);
    ext['k'] = psi_shuffled.dimension(2);
    ext['l'] = mpo.dimension(0);
    ext['m'] = mpo.dimension(1);
    ext['n'] = mpo.dimension(3);
    ext['o'] = mpo.dimension(0);
    ext['p'] = mpo.dimension(1);
    ext['q'] = mpo.dimension(3);
    ext['r'] = envL.dimension(1);
    ext['s'] = envR.dimension(1);

    tools::log->trace("Define tensors for contracting psi and envL");

    Meta<Scalar> cu_psi(psi_shuffled, {'i', 'j', 'k'});
    Meta<Scalar> cu_envL(envL, {'i', 'r', 'l', 'o'});
    Meta<Scalar> cu_psi_envL({'j','k','r','l','o'}, {ext['j'],ext['k'],ext['r'],ext['l'],ext['o']});
    tools::log->trace("Copy to device");
    cu_psi.copyToDevice();
    cu_envL.copyToDevice();
    cu_psi_envL.copyToDevice();

    tools::log->trace("Contract psi and envL");
    cuTensorContract(cu_psi_envL,cu_psi,cu_envL);

    cu_psi.free();
    cu_envL.free();

    tools::log->trace("Define tensors for contracting psi_envL and mpo1");
    Meta<Scalar> cu_mpo1(mpo, {'l', 'm', 'j', 'n'});
    Meta<Scalar> cu_psi_envL_mpo1({'k','r','o','m','n'}, {ext['k'],ext['r'],ext['o'],ext['m'],ext['n']});
    tools::log->trace("Copy to device");
    cu_mpo1.copyToDevice();
    cu_psi_envL_mpo1.copyToDevice();

    tools::log->trace("Contract psi_envL and mpo1");
    cuTensorContract(cu_psi_envL_mpo1,cu_psi_envL,cu_mpo1);

    cu_mpo1.free();
    cu_psi_envL.free();

    tools::log->trace("Define tensors for contracting psi_envL_mpo1 and mpo2");
    Meta<Scalar> cu_mpo2(mpo, {'o', 'p', 'n', 'q'});
    Meta<Scalar> cu_psi_envL_mpo1_mpo2({'k','r','m','p','q'}, {ext['k'],ext['r'],ext['m'],ext['p'],ext['q']});
    tools::log->trace("Copy to device");
    cu_mpo2.copyToDevice();
    cu_psi_envL_mpo1_mpo2.copyToDevice();

    tools::log->trace("Contract psi_envL_mpo1 and mpo2");
    cuTensorContract(cu_psi_envL_mpo1_mpo2,cu_psi_envL_mpo1,cu_mpo2);

    cu_mpo2.free();
    cu_psi_envL_mpo1.free();

    tools::log->trace("Define tensors for contracting psi_envL_mpo1 and mpo2");
    Meta<Scalar> cu_envR(envR, {'k', 's', 'm', 'p'});
    Meta<Scalar> cu_ham_psi_sq({'q','r','s'}, {ext['q'],ext['r'],ext['s']});
    tools::log->trace("Copy to device");
    cu_envR.copyToDevice();
    cu_ham_psi_sq.copyToDevice();

    tools::log->trace("Contract psi_envL_mpo1_mpo2 and envR");
    cuTensorContract(cu_ham_psi_sq,cu_psi_envL_mpo1_mpo2,cu_envR);

    cu_envR.free();
    cu_psi_envL_mpo1_mpo2.free();

    cu_ham_psi_sq.copyFromDevice(ham_sq_psi.data());
    cu_ham_psi_sq.free();
    tools::prof::t_ham_sq_psi_cute->toc();

#if !defined(NDEBUG)
    Eigen::Tensor<Scalar, 3> ham_sq_psi_cpu(dsizes);
    ham_sq_psi_cpu.device(*Textra::omp::dev) =
        psi_shuffled
            .contract(envL , Textra::idx(Textra::array1{0}, Textra::array1{0}))
            .contract(mpo  , Textra::idx(Textra::array2{0, 3}, Textra::array2{2, 0}))
            .contract(mpo  , Textra::idx(Textra::array2{4, 2}, Textra::array2{2, 0}))
            .contract(envR,  Textra::idx(Textra::array3{0, 2, 3}, Textra::array3{0, 2, 3}))
            .shuffle(Textra::array3{1, 0, 2});

    for (long i = 0; i < ham_sq_psi_cpu.size(); i++) {
        if (std::abs(ham_sq_psi_cpu(i) - ham_sq_psi(i))/std::abs(ham_sq_psi_cpu(i)) > 1e-4) {
            tools::log->error("Tensor mismatch > 1e-2 at index {:5}: cpu {:20.16f} != gpu {:20.16f}",i,ham_sq_psi_cpu(i) ,ham_sq_psi(i));
//            throw std::runtime_error("Tensor mismatch > 1e-4 at index "+ std::to_string(i)
//                                     + " " + std::to_string(ham_sq_psi_cpu(i)) + " " + std::to_string(ham_sq_psi(i)));
        }
//        if (not Eigen::internal::isApprox(ham_sq_psi_cpu(i),  ham_sq_psi(i), 1e-4)) {
//            tools::log->error("Tensor mismatch > 1e-4 at index {:5}: {:.8f} != {.8f}",i,ham_sq_psi_cpu(i) ,ham_sq_psi(i));
//            throw std::runtime_error("Tensor approx mismatch > 1e-4 at index "+ std::to_string(i)
//                                     + " " + std::to_string(ham_sq_psi_cpu(i)) + " " + std::to_string(ham_sq_psi(i)));
//        }

    }

#endif


    return std::make_pair(ham_sq_psi,get_ops_m(dsizes[0],dsizes[1],dsizes[2],mpo.dimension(0)));
}

using cplx = std::complex<double>;
using fp32 = float;
using fp64 = double;

// template contract::ResultType<cplx> contract::hamiltonian_squared_dot_psi_cute(const Eigen::Tensor<cplx, 3> &psi_in, const Eigen::Tensor<cplx, 4> &mpo,
//                                                                           const Eigen::Tensor<cplx, 4> &envL, const Eigen::Tensor<cplx, 4> &envR);
template contract::ResultType<fp32> contract::hamiltonian_squared_dot_psi_cute(const Eigen::Tensor<fp32, 3> &psi_in, const Eigen::Tensor<fp32, 4> &mpo,
                                                                           const Eigen::Tensor<fp32, 4> &envL, const Eigen::Tensor<fp32, 4> &envR);
template contract::ResultType<fp64> contract::hamiltonian_squared_dot_psi_cute(const Eigen::Tensor<fp64, 3> &psi_in, const Eigen::Tensor<fp64, 4> &mpo,
                                                                           const Eigen::Tensor<fp64, 4> &envL, const Eigen::Tensor<fp64, 4> &envR);
#endif